
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (2048 * 10000)
#define THREADS_PER_BLOCK 256     
#define num_t float

// The kernel - DOT PRODUCT
__global__ void dot(num_t *a, num_t *b, num_t *c) 
{
  __shared__ num_t temp[THREADS_PER_BLOCK];
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  temp[threadIdx.x] = a[index] * b[index];
  //Synch threads
  __syncthreads();
  if (0 == threadIdx.x) {
    num_t sum = 0.00;
    int i;
    for (i=0; i<THREADS_PER_BLOCK; i++)
      sum += temp[i];
    atomicAdd(c, sum);        
  }
}


// Initialize the vectors:
void init_vector(num_t *x)
{
  int i;
  for (i=0 ; i<N ; i++){
    x[i] = i % 8 + 1;
  }
}

// MAIN
int main(void)
{
  num_t *a, *b, *c;
  num_t *dev_a, *dev_b, *dev_c;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  size_t size = N * sizeof(num_t);

  hipMalloc((void**)&dev_a, size);
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_c, size);

  num_t h_c = 0.0f;
  hipMemcpy(dev_c, &h_c, sizeof(num_t), hipMemcpyHostToDevice);

  a = (num_t*)malloc(size);
  b = (num_t*)malloc(size);
  c = (num_t*)malloc(size);

  init_vector(a);
  init_vector(b);

  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  hipEventRecord(start);
  dot<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_c);
  hipEventRecord(stop);

  hipMemcpy(c, dev_c, sizeof(num_t), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop); 
  float milliseconds = 0;
  hipEventElapsedTime (&milliseconds, start, stop);
  //printf("Kernal Execution Time: %d\n", milliseconds);
  //printf("Effective Bandwidth (GB/s): %f", N*4*3/milliseconds/1e6);
  printf("Vector Size: %d\n", N);
  printf("Execution Time: %fms\n", milliseconds);
  printf("Inner Product: %f\n", *c);

  free(a); free(b); free(c);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

}
